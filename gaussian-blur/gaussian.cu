#include "hip/hip_runtime.h"
// Josh Shepherd 1700471
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"

/**
Applies Gaussian Blur using CUDA. 
*/
__global__
void gaussianBlur(unsigned char* deviceInput, unsigned char* deviceOutput) {
	/// Get unique id of for the thread
	int uid = blockDim.x * blockIdx.x + threadIdx;

	printf("Thread ID: '%d'\n", uid);
}

/**
	Gaussian blur using CUDA threads. Takes two arguments, 
	1: Path name the input png file
	2: Path name of the gaussian blurred png file
*/
int main (int argc, char* argv[]) {
	// Get file name of png
	char* fileName = "img.png";
    if (argc > 1)
        fileName = argv[1];
    // Get gaussian blur output file name
    char* outputFileName = "output.png";
    if (argc > 2)
    	outputFileName = argv[2];

	printf("Blurring image '%s'\n", fileName);
   
	/// Initially load PNG file using lodepng
	unsigned int width, height;
	unsigned char* pngValues;
	lodepng_decode32_file(&pngValues, &width, &height, fileName);

	// Check if image loaded is valid
	if (width <= 0 || height <= 0) {
        printf("Unable to decode image. Validate file and try again");
        exit(-1);
    }
    
    /// Malloc device original png values
	unsigned char* d_originalVals;
    int originalValsLength = width * height;	
    hipMalloc((void**) &d_originalVals, originalValsLength);
    /// Transfer from CPU to GPU
    hipMemcpy(d_originalVals, pngValues, originalValsLength, hipMemcpyHostToDevice);
    
    // cuda malloc the final blurred vals array using width * height
    unsigned char* d_blurredVals;
    hipMalloc((void**) &d_blurredVals, blurredArrayLength);
   

    /// Launch CUDA to gaussian blur original vals to blurred vals
    gaussianBlur<<< dim3(1, 1, 1), dim3(1, 1, 1) >>>(d_originalVals, d_blurredVals);
   	hipDeviceSynchronize();
    
    
    /// Copy final CUDA blurred img vals to CPU
    unsigned char* blurredImgVals;
    hipMemcpy(blurredImgVals, d_blurredVals, originalValsLength, hipMemcpyDeviceToHost);
    
    /// Save blurred values to png file
    unsigned char* threadImgValues;
    lodepng_encode32_file(outFileName, threadImgValues, width, height);
    printf("Successfully blurred the image into ./'%s'\n", outFileName);
    
    /// Free any malloc & CUDA malloc
    free(pngValues);
    free(blurredImgVals);
    hipFree(d_blurredVals);
    hipFree(d_originalVals);
}
